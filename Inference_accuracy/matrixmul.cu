#include "hip/hip_runtime.h"
//四种优化后的ASCM
//w
#include <stdlib.h>
#include <malloc.h>
#include <stdint.h>
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include "stdint.h"
#include "math_functions.h"

extern "C" {
#include "gemm.h"
#include "hip/hip_runtime.h"
}  


 //#define __DEBUG  //通过定义DEBUG来决定要不要输出中间结果

#ifdef __DEBUG
#define DEBUG(format, ...) printf(format, ##__VA_ARGS__)
#else
#define DEBUG(format, ...)
#endif

// typedef uint8_t elemtype;

// #define SIZE 256              // 输入的量化值的位数
// #define bit_w 8
            //量化值实际需要用来表示的位数

__device__ int scale;
 __device__ int b_bit1;
// __device__ int WeightNumerator;           //weight分子
// __device__ int ActivationNumerator;        //act分子
// __device__ int weightPartition;
// __device__ int activationPartition;
// __device__ int addPartition;                //大行+大列

__device__ int StochasticComputing_6(double weight,double activation, int bit_w, int bit_w2){
	int Numerator1, Numerator2;
	int size = powf(2, bit_w);
  int size1 = powf(2, bit_w2);
	int part_w = bit_w / 2;
	int bn1_h, bn1_l, bn2_h, bn2_l; 
	int H1, H2;
	
	double b1, b2;
	int I1, I2, I3, I4;  //相与结果和取反相与结果
	int R, R2;  //每部分的结果  
	int partition1, partition2; //用来记录边界
	
	//将小数去掉小数点，转换成bit_w位整数，数值大小代表了SN中“1”的个数 
	Numerator1 = (int)round(weight * size);
	Numerator2 = (int)round(activation * size1);
	// printf("%d %d %d\t%d\n", size, size1, Numerator1, Numerator2);
	// printf("%f\t%d\n", activation, Numerator2);

	
		//始终让 Numerator1是硬连线， Numerator2是集中分布 
		//*****************9.23修改**************
	  if((Numerator1 <= Numerator2) && bit_w == bit_w2){
		 	  int temp = Numerator1;
		    Numerator1 = Numerator2;
		    Numerator2 = temp;
		}
		//printf("%d %d \n", Numerator1, Numerator2);
		//分别获取两个数的四个部分 

    if(bit_w == 6){
			bn1_h = (Numerator1 & 0x38) >> (part_w);
			bn1_l = (Numerator1 & 0x07);
			bn2_h = (Numerator2 & 0x38) >> (part_w);
			bn2_l = (Numerator2 & 0x07);
		}
    if(bit_w == 3){
			bn1_h = (Numerator1 & 0x4) >> (bit_w-part_w);
			bn1_l = (Numerator1 & 0x3);
			bn2_h = (Numerator2 & 0x6) >> (part_w);
			bn2_l = (Numerator2 & 0x1);
		}

		
		H1 = bn1_h;
		H2 = bn2_h;
		
     
    if(bit_w == 6){
    
      if (bn1_h >= 0 && bn1_h < 1){
				I1 = 0;
			}else if (bn1_h >= 1 && bn1_h <2){
				I1 = bn2_l / 4;
			}else if (bn1_h >= 2 && bn1_h < 3){
				I1 = bn2_l / 4;
			}else if (bn1_h >= 3 && bn1_h < 4){
				I1 = bn2_l / 2;
			}else if (bn1_h >= 4 && bn1_h < 5){
				I1 = bn2_l / 2;
			}else if (bn1_h >= 5 && bn1_h < 6){
				I1 = bn2_l * 0.75;
			}else if (bn1_h >= 6 && bn1_h < 7){
				I1 = bn2_l * 0.75;
			}else{
				I1 = bn2_l;
			}
      
      if (bn1_l >= 0 && bn1_l < 1){
				I2 = 0;
			}else if (bn1_l >= 1 && bn1_l < 2){
				I2 = bn2_h / 4;
			}else if (bn1_l >= 2 && bn1_l < 3){
				I2 = bn2_h / 4;
			}else if (bn1_l >= 3 && bn1_l < 4){
				I2 = bn2_h / 2;
			}else if (bn1_l >= 4 && bn1_l < 5){
				I2 = bn2_h / 2;
			}else if (bn1_l >= 5 && bn1_l < 6){
				I2 = bn2_h *0.75;
			}else if (bn1_l >= 6 && bn1_l < 7){
				I2 = bn2_h * 0.75;
			}else{
				I2 = bn2_h;
			}	
    	
    }
    
    if(bit_w == 3){
    
       if (bn1_h == 1){
          I1 = bn2_l;
       }else{
          I1 = 0;
       }
       
       if (bn1_l > 2)   //修改后w
			 {
				 I2 = bn2_h;
			 }else{
         if(bn1_l == 0){
            I2 = 0;
         }else{
            I2 = bn2_h / 2;
         }
			 }	
    }
    

    
		//printf("%d %d %d %d    %d %d %d \n", bn1_h,bn1_l,bn2_h,bn2_l,part_w,I1,I2);
    //printf("%d %d %d \n",part_w,I1,I2);
		
		R = I1 + I2;
		
    //printf("%d %d H:%d %d\n",part_w,bit_w-part_w,H1,H2);
    //printf("%d %d H:%d %d\n",bit_w,bit_w2,H1,H2);
		//2位的操作就直接给个逻辑 
		if(part_w > 2 || (bit_w2-part_w) > 2){
			b1 = (double)H1 / (double)powf(2, part_w);
			b2 = (double)H2 / (double)powf(2, bit_w2-part_w);
      R2 = StochasticComputing_6(b1, b2, part_w, bit_w2-part_w);
      
			//printf("%d %d \n", part_w, bit_w-part_w);

			//迭代的结果需要扩展！！！！！
			R2 = R2 << part_w;
      //printf("part_w:%d %d %d \n",part_w, R, R2);
      //
			R = R + R2;
		}	
		
		return R;	
}



__device__ int StochasticComputing_7(double weight,double activation, int bit_w, int bit_w2){
	int Numerator1, Numerator2;
	int size = powf(2, bit_w);
  int size1 = powf(2, bit_w2);
	int part_w = bit_w / 2;
	int bn1_h, bn1_l, bn2_h, bn2_l; 
	int H1, H2;
	
	double b1, b2;
	int I1, I2, I3, I4;  //相与结果和取反相与结果
	int R, R2;  //每部分的结果  
	int partition1, partition2; //用来记录边界
	
	//将小数去掉小数点，转换成bit_w位整数，数值大小代表了SN中“1”的个数 
	Numerator1 = (int)round(weight * size);
	Numerator2 = (int)round(activation * size1);
	// printf("%d %d %d\t%d\n", size, size1, Numerator1, Numerator2);
	// printf("%f\t%d\n", activation, Numerator2);

	
		//始终让 Numerator1是硬连线， Numerator2是集中分布 
		//*****************9.23修改**************
	  if((Numerator1 <= Numerator2) && bit_w == bit_w2){
		 	  int temp = Numerator1;
		    Numerator1 = Numerator2;
		    Numerator2 = temp;
		}
		//printf("%d %d \n", Numerator1, Numerator2);
   
		//分别获取两个数的四个部分 
    
    if(bit_w == 7){
			bn1_h = (Numerator1 & 0x70) >> (bit_w-part_w);
			bn1_l = (Numerator1 & 0x0F);
			bn2_h = (Numerator2 & 0x78) >> (part_w);
			bn2_l = (Numerator2 & 0x07);
		}
   
    if(bit_w == 3){
			bn1_h = (Numerator1 & 0x4) >> (bit_w-part_w);
			bn1_l = (Numerator1 & 0x3);
			bn2_h = (Numerator2 & 0xC) >> (bit_w-part_w);
			bn2_l = (Numerator2 & 0x3);
      //printf("%d %d afdfa: %d %d %d %d\n", Numerator1,Numerator2,bn1_h,bn1_l,bn2_h,bn2_l);
    }

		H1 = bn1_h;
		H2 = bn2_h;
    
    if(bit_w == 7){
    
      if (bn1_h >= 0 && bn1_h < 1){
				I1 = 0;
			}else if (bn1_h >= 1 && bn1_h <2){
				I1 = bn2_l / 4;
			}else if (bn1_h >= 2 && bn1_h < 3){
				I1 = bn2_l / 4;
			}else if (bn1_h >= 3 && bn1_h < 4){
				I1 = bn2_l / 2;
			}else if (bn1_h >= 4 && bn1_h < 5){
				I1 = bn2_l / 2;
			}else if (bn1_h >= 5 && bn1_h < 6){
				I1 = bn2_l * 0.75;
			}else if (bn1_h >= 6 && bn1_h < 7){
				I1 = bn2_l * 0.75;
			}else{
				I1 = bn2_l;
			}
      
      if (bn1_l >= 0 && bn1_l < 2){
				I2 = 0;
			}else if (bn1_l >= 2 && bn1_l < 4){
				I2 = bn2_h / 4;
			}else if (bn1_l >= 4 && bn1_l < 6){
				I2 = bn2_h / 4;
			}else if (bn1_l >= 6 && bn1_l < 8){
				I2 = bn2_h / 2;
			}else if (bn1_l >= 8 && bn1_l < 10){
				I2 = bn2_h / 2;
			}else if (bn1_l >= 10 && bn1_l < 12){
				I2 = bn2_h *0.75;
			}else if (bn1_l >= 12 && bn1_l < 14){
				I2 = bn2_h * 0.75;
			}else{
				I2 = bn2_h;
			}	
    
    }	
    
    if(bit_w == 3){
    
       if (bn1_h == 1){
          I1 = bn2_l;
       }else{
          I1 = 0;
       }
       
       if (bn1_l > 2)   //修改后w
			 {
				 I2 = bn2_h;
			 }else{
         if(bn1_l == 0){
            I2 = 0;
         }else{
            I2 = bn2_h / 2;
         }
			 }	
    }
    
		//printf("%d %d %d %d    %d %d %d \n", bn1_h,bn1_l,bn2_h,bn2_l,part_w,I1,I2);
    //printf("%d %d %d \n",part_w,I1,I2);
		
		R = I1 + I2;
		
    //printf("%d %d H:%d %d\n",part_w,bit_w-part_w,H1,H2);
    //printf("%d %d H:%d %d\n",bit_w,bit_w2,H1,H2);
		//2位的操作就直接给个逻辑 
		if(part_w > 2){
			b1 = (double)H1 / (double)powf(2, part_w);
			b2 = (double)H2 / (double)powf(2, bit_w2-part_w);
      R2 = StochasticComputing_7(b1, b2, part_w, bit_w2-part_w);
      //R2=1;
			//printf("%d %d \n", part_w, bit_w-part_w);

			//迭代的结果需要扩展！！！！！
			R2 = R2 << part_w;
      //printf("part_w:%d %d %d \n",part_w, R, R2);
      //
			R = R + R2;
		}	
		
		return R;	
}

__device__ int StochasticComputing_8(double weight,double activation, int bit_w, int bit_w2){
	int Numerator1, Numerator2;
	int size = powf(2, bit_w);
  int size1 = powf(2, bit_w2);
	int part_w = bit_w / 2;
	int bn1_h, bn1_l, bn2_h, bn2_l; 
	int H1, H2;
	
	double b1, b2;
	int I1, I2, I3, I4;  //相与结果和取反相与结果
	int R, R2;  //每部分的结果  
	int partition1, partition2; //用来记录边界
	
	//将小数去掉小数点，转换成bit_w位整数，数值大小代表了SN中“1”的个数 
	Numerator1 = (int)round(weight * size);
	Numerator2 = (int)round(activation * size1);
	// printf("%d %d %d\t%d\n", size, size1, Numerator1, Numerator2);
	// printf("%f\t%d\n", activation, Numerator2);

	
		//始终让 Numerator1是硬连线， Numerator2是集中分布 
		//*****************9.23修改**************
	  if((Numerator1 <= Numerator2) && bit_w == bit_w2){
		 	  int temp = Numerator1;
		    Numerator1 = Numerator2;
		    Numerator2 = temp;
		}
		//printf("%d %d \n", Numerator1, Numerator2);
		//分别获取两个数的四个部分 

		if(bit_w == 8){
			bn1_h = (Numerator1 & 0xF0) >> (part_w);
			bn1_l = (Numerator1 & 0x0F);
			bn2_h = (Numerator2 & 0xF0) >> (part_w);
			bn2_l = (Numerator2 & 0x0F);
		}
		if(bit_w == 4){
			bn1_h = (Numerator1 & 0xC) >> (part_w);
			bn1_l = (Numerator1 & 0x3);
			bn2_h = (Numerator2 & 0xC) >> (part_w);
			bn2_l = (Numerator2 & 0x3);
		}

		
		H1 = bn1_h;
		H2 = bn2_h; 

		if(bit_w == 8){
			//*****************9.23修改**************
			// if(bn2_l > bn1_h) {
			// int temp = bn2_l;
			// bn2_l = bn1_h;
			// bn1_h = temp;
			// }
			if (bn1_h >= 0 && bn1_h < 2)
			{
				I1 = 0;
			}else if (bn1_h >= 2 && bn1_h <4){
				I1 = bn2_l / 4;
			}else if (bn1_h >= 4 && bn1_h < 6){
				I1 = bn2_l / 4;
			}else if (bn1_h >= 6 && bn1_h < 8){
				I1 = bn2_l / 2;
			}else if (bn1_h >= 8 && bn1_h < 10){
				I1 = bn2_l / 2;
			}else if (bn1_h >= 10 && bn1_h < 12){
				I1 = bn2_l * 0.75;
			}else if (bn1_h >= 12 && bn1_h < 14){
				I1 = bn2_l * 0.75;
			}else{
				I1 = bn2_l;
			}

			if (bn1_l >= 0 && bn1_l < 2)
			{
				I2 = 0;
			}else if (bn1_l >= 2 && bn1_l < 4){
				I2 = bn2_h / 4;
			}else if (bn1_l >= 4 && bn1_l < 6){
				I2 = bn2_h / 4;
			}else if (bn1_l >= 6 && bn1_l < 8){
				I2 = bn2_h / 2;
			}else if (bn1_l >= 8 && bn1_l < 10){
				I2 = bn2_h / 2;
			}else if (bn1_l >= 10 && bn1_l < 12){
				I2 = bn2_h *0.75;
			}else if (bn1_l >= 12 && bn1_l < 14){
				I2 = bn2_h * 0.75;
			}else{
				I2 = bn2_h;
			}	
		}

		if(bit_w == 4){

			if (bn1_h > 2)    //修改后w
			{
				I1 = bn2_l;
			}else{
        if(bn1_h == 0){
            I1 = 0;
        }else{
            I1 = bn2_l / 2;
        }
			}

			if (bn1_l > 2)   //修改后w
			{
				I2 = bn2_h;
			}else{
        if(bn1_l == 0){
            I2 = 0;
        }else{
            I2 = bn2_h / 2;
        }
			}	
		}

    
		//printf("%d %d %d %d    %d %d %d \n", bn1_h,bn1_l,bn2_h,bn2_l,part_w,I1,I2);
    //printf("%d %d %d \n",part_w,I1,I2);
		
		R = I1 + I2;
		
    //printf("%d %d H:%d %d\n",part_w,bit_w-part_w,H1,H2);
    //printf("%d %d H:%d %d\n",bit_w,bit_w2,H1,H2);
		//2位的操作就直接给个逻辑 
		if(part_w > 2 || (bit_w2-part_w) > 2){
			b1 = (double)H1 / (double)powf(2, part_w);
			b2 = (double)H2 / (double)powf(2, bit_w2-part_w);
      R2 = StochasticComputing_8(b1, b2, part_w, bit_w2-part_w);
      
			//printf("%d %d \n", part_w, bit_w-part_w);

			//迭代的结果需要扩展！！！！！
			R2 = R2 << part_w;
      //printf("part_w:%d %d %d \n",part_w, R, R2);
      //
			R = R + R2;
		}	
		
		return R;	
}

__device__ int StochasticComputing_9(double weight,double activation, int bit_w, int bit_w2){
	int Numerator1, Numerator2;
	int size = powf(2, bit_w);
  int size1 = powf(2, bit_w2);
	int part_w = bit_w / 2;
	int bn1_h, bn1_l, bn2_h, bn2_l; 
	int H1, H2;
	
	double b1, b2;
	int I1, I2, I3, I4;  //相与结果和取反相与结果
	int R, R2;  //每部分的结果  
	int partition1, partition2; //用来记录边界
	
	//将小数去掉小数点，转换成bit_w位整数，数值大小代表了SN中“1”的个数 
	Numerator1 = (int)round(weight * size);
	Numerator2 = (int)round(activation * size1);
	// printf("%d %d %d\t%d\n", size, size1, Numerator1, Numerator2);
	// printf("%f\t%d\n", activation, Numerator2);

	
		//始终让 Numerator1是硬连线， Numerator2是集中分布 
		//*****************9.23修改**************
	  if((Numerator1 <= Numerator2) && bit_w == bit_w2){
		 	  int temp = Numerator1;
		    Numerator1 = Numerator2;
		    Numerator2 = temp;
		}
		//printf("%d %d \n", Numerator1, Numerator2);
		//分别获取两个数的四个部分 
    if(bit_w == 9){
			bn1_h = (Numerator1 & 0x1E0) >> (bit_w-part_w);
			bn1_l = (Numerator1 & 0x01F);
			bn2_h = (Numerator2 & 0x1F0) >> (part_w);
			bn2_l = (Numerator2 & 0x00F);
		}
    if(bit_w == 2){
			bn1_h = (Numerator1 & 0x2) >> (part_w);
			bn1_l = (Numerator1 & 0x1);
			bn2_h = (Numerator2 & 0x6) >> (part_w);
			bn2_l = (Numerator2 & 0x1);
      //printf("%d %d afdfa: %d %d %d %d\n", Numerator1,Numerator2,bn1_h,bn1_l,bn2_h,bn2_l);
		}
    if(bit_w == 4){
			bn1_h = (Numerator1 & 0xC) >> (part_w);
			bn1_l = (Numerator1 & 0x3);
			bn2_h = (Numerator2 & 0x1C) >> (part_w);
			bn2_l = (Numerator2 & 0x3);
      //printf("%d %d %d %d afdfa: %d %d %d %d\n", bit_w, bit_w2, Numerator1,Numerator2,bn1_h,bn1_l,bn2_h,bn2_l);
		}
		
		H1 = bn1_h;
		H2 = bn2_h;
		
   
   if(bit_w == 9){
			
			if (bn1_h >= 0 && bn1_h < 2)
			{
				I1 = 0;
			}else if (bn1_h >= 2 && bn1_h <4){
				I1 = bn2_l / 4;
			}else if (bn1_h >= 4 && bn1_h < 6){
				I1 = bn2_l / 4;
			}else if (bn1_h >= 6 && bn1_h < 8){
				I1 = bn2_l / 2;
			}else if (bn1_h >= 8 && bn1_h < 10){
				I1 = bn2_l / 2;
			}else if (bn1_h >= 10 && bn1_h < 12){
				I1 = bn2_l * 0.75;
			}else if (bn1_h >= 12 && bn1_h < 14){
				I1 = bn2_l * 0.75;
			}else{
				I1 = bn2_l;
			}

			if (bn1_l >= 0 && bn1_l < 2)
			{
				I2 = 0;
			}else if (bn1_l >= 2 && bn1_l < 6){
				I2 = bn2_h / 8;
			}else if (bn1_l >= 6 && bn1_l < 10){
				I2 = bn2_h / 4;
			}else if (bn1_l >= 10 && bn1_l < 14){
				I2 = bn2_h * 0.375;
			}else if (bn1_l >= 14 && bn1_l < 18){
				I2 = bn2_h / 2;
			}else if (bn1_l >= 18 && bn1_l < 22){
				I2 = bn2_h *0.625;
			}else if (bn1_l >= 22 && bn1_l < 26){
				I2 = bn2_h * 0.75;
			}else if (bn1_l >= 26 && bn1_l < 30){
				I2 = bn2_h * 0.875;
			}else{
				I2 = bn2_h;
			}	
		}

		if(bit_w == 4){
			if (bn1_h > 2)    //修改后w
			{
				I1 = bn2_l;
			}else{
        if(bn1_h == 0){
            I1 = 0;
        }else{
            I1 = bn2_l / 2;
        }
			}

			if (bn1_l > 2)   //修改后w
			{
				I2 = bn2_h;
			}else{
        if(bn1_l == 0){
            I2 = 0;
        }else{
            I2 = bn2_h / 2;
        }
			}	
		}
    
    if(bit_w == 2){
    
       if (bn1_h == 1){
          I1 = bn2_l;
       }else{
          I1 = 0;
       }
       
       if (bn1_l == 1)   
			 {
				 I2 = bn2_h;
			 }else{
         I2 = 0;
			 }	
    }
    
		//printf("%d %d %d %d    %d %d %d \n", bn1_h,bn1_l,bn2_h,bn2_l,part_w,I1,I2);
    //printf("%d %d %d \n",part_w,I1,I2);
		
		R = I1 + I2;
		
    //printf("%d %d H:%d %d\n",part_w,bit_w-part_w,H1,H2);
    //printf("%d %d H:%d %d\n",bit_w,bit_w2,H1,H2);
		//2位的操作就直接给个逻辑 
		if(part_w > 2 || (bit_w2-part_w) > 2){
			b1 = (double)H1 / (double)powf(2, part_w);
			b2 = (double)H2 / (double)powf(2, bit_w2-part_w);
      R2 = StochasticComputing_9(b1, b2, part_w, bit_w2-part_w);
      
			//printf("%d %d \n", part_w, bit_w-part_w);

			//迭代的结果需要扩展！！！！！
			R2 = R2 << part_w;
      //printf("part_w:%d %d %d \n",part_w, R, R2);
      //
			R = R + R2;
		}	
		
		return R;	
}

__device__ int StochasticComputing_10(double weight,double activation, int bit_w, int bit_w2){
	int Numerator1, Numerator2;
	int size = powf(2, bit_w);
  int size1 = powf(2, bit_w2);
	int part_w = bit_w / 2;
	int bn1_h, bn1_l, bn2_h, bn2_l; 
	int H1, H2;
	
	double b1, b2;
	int I1, I2, I3, I4;  //相与结果和取反相与结果
	int R, R2;  //每部分的结果  
	int partition1, partition2; //用来记录边界
	
	//将小数去掉小数点，转换成bit_w位整数，数值大小代表了SN中“1”的个数 
	Numerator1 = (int)round(weight * size);
	Numerator2 = (int)round(activation * size1);
	// printf("%d %d %d\t%d\n", size, size1, Numerator1, Numerator2);
	// printf("%f\t%d\n", activation, Numerator2);

	
		//始终让 Numerator1是硬连线， Numerator2是集中分布 
		//*****************9.23修改**************
	  if((Numerator1 <= Numerator2) && bit_w == bit_w2){
		 	  int temp = Numerator1;
		    Numerator1 = Numerator2;
		    Numerator2 = temp;
		}
		//printf("%d %d \n", Numerator1, Numerator2);
		//分别获取两个数的四个部分 
    if(bit_w == 10){
			bn1_h = (Numerator1 & 0x3E0) >> (part_w);
			bn1_l = (Numerator1 & 0x01F);
			bn2_h = (Numerator2 & 0x3E0) >> (part_w);
			bn2_l = (Numerator2 & 0x01F);
		}
    if(bit_w == 2){
			bn1_h = (Numerator1 & 0x2) >> (part_w);
			bn1_l = (Numerator1 & 0x1);
			bn2_h = (Numerator2 & 0x6) >> (part_w);
			bn2_l = (Numerator2 & 0x1);
      //printf("%d %d afdfa: %d %d %d %d\n", Numerator1,Numerator2,bn1_h,bn1_l,bn2_h,bn2_l);
		}
    if(bit_w == 5){
			bn1_h = (Numerator1 & 0x18) >> (bit_w-part_w);
			bn1_l = (Numerator1 & 0x07);
			bn2_h = (Numerator2 & 0x1c) >> (part_w);
			bn2_l = (Numerator2 & 0x03);
		}
		
		H1 = bn1_h;
		H2 = bn2_h;
		
		
   
   if(bit_w == 10){
			
			if (bn1_h >= 0 && bn1_h < 2)
			{
				I1 = 0;
			}else if (bn1_h >= 2 && bn1_h < 6){
				I1 = bn2_l / 8;
			}else if (bn1_h >= 6 && bn1_h < 10){
				I1 = bn2_l / 4;
			}else if (bn1_h >= 10 && bn1_h < 14){
				I1 = bn2_l * 0.375;
			}else if (bn1_h >= 14 && bn1_h < 18){
				I1 = bn2_l / 2;
			}else if (bn1_h >= 18 && bn1_h < 22){
				I1 = bn2_l *0.625;
			}else if (bn1_h >= 22 && bn1_h < 26){
				I1 = bn2_l * 0.75;
			}else if (bn1_h >= 26 && bn1_h < 30){
				I1 = bn2_l * 0.875;
			}else{
				I1 = bn2_l;
			}	

			if (bn1_l >= 0 && bn1_l < 2)
			{
				I2 = 0;
			}else if (bn1_l >= 2 && bn1_l < 6){
				I2 = bn2_h / 8;
			}else if (bn1_l >= 6 && bn1_l < 10){
				I2 = bn2_h / 4;
			}else if (bn1_l >= 10 && bn1_l < 14){
				I2 = bn2_h * 0.375;
			}else if (bn1_l >= 14 && bn1_l < 18){
				I2 = bn2_h / 2;
			}else if (bn1_l >= 18 && bn1_l < 22){
				I2 = bn2_h *0.625;
			}else if (bn1_l >= 22 && bn1_l < 26){
				I2 = bn2_h * 0.75;
			}else if (bn1_l >= 26 && bn1_l < 30){
				I2 = bn2_h * 0.875;
			}else{
				I2 = bn2_h;
			}	
		}
   
    if(bit_w == 5){
		
			if (bn1_h > 2){   
				I1 = bn2_l;
			}else{
        if(bn1_h == 0){
            I1 = 0;
        }else{
            I1 = bn2_l / 2;
        }
			}

			if (bn1_l >= 0 && bn1_l < 1){
				I2 = 0;
			}else if (bn1_l >= 1 && bn1_l < 2){
				I2 = bn2_h / 4;
			}else if (bn1_l >= 2 && bn1_l < 3){
				I2 = bn2_h / 4;
			}else if (bn1_l >= 3 && bn1_l < 4){
				I2 = bn2_h / 2;
			}else if (bn1_l >= 4 && bn1_l < 5){
				I2 = bn2_h / 2;
			}else if (bn1_l >= 5 && bn1_l < 6){
				I2 = bn2_h *0.75;
			}else if (bn1_l >= 6 && bn1_l < 7){
				I2 = bn2_h * 0.75;
			}else{
				I2 = bn2_h;
			}	
		}
    
    if(bit_w == 2){
    
       if (bn1_h == 1){
          I1 = bn2_l;
       }else{
          I1 = 0;
       }
       
       if (bn1_l == 1)   
			 {
				 I2 = bn2_h;
			 }else{
         I2 = 0;
			 }	
    }
    
		//printf("%d %d %d %d    %d %d %d \n", bn1_h,bn1_l,bn2_h,bn2_l,part_w,I1,I2);
    //printf("%d %d %d \n",part_w,I1,I2);
		
		R = I1 + I2;
		
    //printf("%d %d H:%d %d\n",part_w,bit_w-part_w,H1,H2);
    //printf("%d %d H:%d %d\n",bit_w,bit_w2,H1,H2);
		//2位的操作就直接给个逻辑 
		if(part_w > 2 || (bit_w2-part_w) > 2){
			b1 = (double)H1 / (double)powf(2, part_w);
			b2 = (double)H2 / (double)powf(2, bit_w2-part_w);
      R2 = StochasticComputing_10(b1, b2, part_w, bit_w2-part_w);
      
			//printf("%d %d \n", part_w, bit_w-part_w);

			//迭代的结果需要扩展！！！！！
			R2 = R2 << part_w;
      //printf("part_w:%d %d %d \n",part_w, R, R2);
      //
			R = R + R2;
		}	
		
		return R;	
}


//高两位相乘，也就是最内层递归的结果，直接给逻辑
__device__ int operTwo(double weight,double activation){
	
	int size = powf(2, b_bit1);
	int Numerator1 = (int)round(weight * size);
	int Numerator2 = (int)round(activation * size);
	
	int b1_h_2;
	int b2_h_2;  //取两个数的高两位
	int result;  //结果 

  if(Numerator1 <= Numerator2){
     int temp = Numerator1;
     Numerator1 = Numerator2;
     Numerator2 = temp;
	}
   
  if(b_bit1 == 8){
     b1_h_2 = Numerator1 >> (b_bit1 - 2);
     b2_h_2 = Numerator2 >> (b_bit1 - 2);
     result = (b1_h_2 * b2_h_2) << (b_bit1 - 4);
  }else{
     if(b_bit1 == 6 || b_bit1 == 7 || b_bit1 == 9 || b_bit1 == 10){
        b1_h_2 = Numerator1 >> (b_bit1 - 1);
        b2_h_2 = Numerator2 >> (b_bit1 - 2);
        result = (b1_h_2 * b2_h_2) << (b_bit1 - 3);
     }
  }
		
		return result;


} 



__device__ double Computing(double weight,double activation){
	int bit_w = b_bit1;
  //double weight = 0.6328125;
  //double activation = 0.78515625;//0.28515625;   0.78515625

//如果有负数
    double flag=1.0;
    if((weight<0 && activation>0) || (weight>0 && activation<0))
        flag=-1.0;
                
    weight=fabs(weight);
    activation=fabs(activation); 
    
    //缩小为二分之一
    if((weight + activation) > 1){
       weight = weight * 0.5;
       activation = activation * 0.5;
       scale = 1;   //记录是否缩放
    }else{
       scale = 0;
    }
    
    
    
// 如果有0/1这些特殊值，就直接输出了，不算了
    if (activation == 0 || weight == 0)
        {
            return 0;
        }
    if (activation == 1)
        {
            return weight;
        }
    if (weight == 1)
        {
            return activation;
        }

    //加上这个if训练出错，降低学习率0.01 ---> 0.001
    if (activation > 1.0 || weight > 1.0)
    {
    	return activation * weight * flag;
    }

    double result;
    
    if(bit_w == 6){
       result= ((StochasticComputing_6(weight, activation, bit_w, bit_w) + operTwo(weight,activation)) * 1.0) / 64;
    }else if(bit_w == 7){
       result= ((StochasticComputing_7(weight, activation, bit_w, bit_w) + operTwo(weight,activation)) * 1.0) / 128;
    }else if(bit_w == 8){
       result= ((StochasticComputing_8(weight, activation, bit_w, bit_w) + operTwo(weight,activation)) * 1.0) / 256;
    }else if(bit_w == 9){
       result= ((StochasticComputing_9(weight, activation, bit_w, bit_w) + operTwo(weight,activation)) * 1.0) / 512;
    }else if(bit_w == 10){
       result= ((StochasticComputing_10(weight, activation, bit_w, bit_w) + operTwo(weight,activation)) * 1.0) / 1024;
    }
    //result = ((StochasticComputing(weight, activation, bit_w, bit_w) + operTwo(weight,activation, bit_w)) * 1.0) / 256;
    
    if(scale == 1){
       result = result * 4;
    }

    //printf("%lf %lf %d %d %d %lf\n", weight, activation, (int)round(weight * 256), (int)round(activation * 256),bit_w,result);
    //printf("%d %d %lf\n", StochasticComputing_10(weight, activation, bit_w, bit_w), operTwo(weight,activation), result);
    //printf("%d %lf\n", bit_w,result);
    
    return result * flag;

}



 
__global__ void MatrixMul_device(int TA, int TB, float ALPHA, float BETA, float *a, int a_rows, int a_cols, float *b, int b_rows, int b_cols, float *c, int b_bit) {
    int tix = threadIdx.x;

    int bix = blockIdx.x;

    int bdx = blockDim.x;
    
    b_bit1 = b_bit;
 
if(TA ==1 && TB ==0){

    //A的转置
    for (int i = tix; i < b_cols; i += bdx) {
	   float sum = 0;
        for (int k = 0; k < a_cols; k++) {
		sum += Computing(ALPHA * a[bix+a_rows*k],b[b_cols*k+i]);			
		//	sum += normal(ALPHA * a[bix+a_rows*k],b[b_cols*k+i]);
			//sum += ALPHA * a[bix+a_rows*k] * b[b_cols*k+i];
			
			
		//	printf("%lf\t%lf\t%lf\n" ,ALPHA * a[bix+a_rows*k],b[b_cols*k+i],ALPHA * a[bix+a_rows*k]*b[b_cols*k+i]);
	//printf("%lf\n" ,ALPHA * a[bix*a_cols+k] * b[k*b_cols+i]);
		// printf("%lf\t%lf\n" ,ALPHA * a[bix*a_cols+k], b[k*b_cols+i]);
        }
        c[bix*b_cols+i] = sum;   
     //   printf("-----------------------%lf----------------------\n\n\n",sum);

      
    } 
    
    } 
    
if(TA == 0 && TB == 1){
    //B的转置
     for (int i = tix; i < b_cols; i += bdx) {
        float sum = 0;
        float aaaaa;
        for (int k = 0; k < a_cols; k++) {
         sum += Computing(ALPHA * a[bix*a_cols+k], b[b_rows*i+k]);
         //sum += ALPHA * a[bix*a_cols+k] * b[b_rows*i+k];
         //printf("%d\n",b_bit1);
         //printf("%lf\t%lf--->%lf\t%lf\n", b[b_rows*i+k], ALPHA * a[bix*a_cols+k], b[b_rows*i+k] * ALPHA * a[bix*a_cols+k], aaaaa);
		//	sum += normal(ALPHA * a[bix*a_cols+k], b[b_rows*i+k]);
		    //sum += ALPHA * a[bix*a_cols+k] * b[b_rows*i+k];
		
	//	printf("%lf\t%lf\t%lf\n" ,ALPHA * a[bix*a_cols+k] , b[b_rows*i+k],ALPHA * a[bix*a_cols+k] * b[b_rows*i+k]);
		
		// printf("%lf\t%lf\n" ,ALPHA * a[bix*a_cols+k], b[k*b_cols+i]);
        }
         c[bix* b_cols+i] = sum;
    //   printf("---------------------%lf-------------------\n\n",sum);
    }
    
    }  
    
 if(TA == 0 && TB == 0){
    //不转置
    for (int i = tix; i < b_cols; i += bdx) {
      float sum = 0;
        for (int k = 0; k < a_cols; k++) {
			sum += Computing(ALPHA * a[bix*a_cols+k] , b[k*b_cols+i]);
      //sum += 5;
	//		sum += normal(ALPHA * a[bix*a_cols+k] , b[k*b_cols+i]);
	//		sum += ALPHA * a[bix*a_cols+k] * b[k*b_cols+i];
	
				// printf("%lf\t%lf\t%lf\n" ,ALPHA * a[bix*a_cols+k] , b[k*b_cols+i],ALPHA * a[bix*a_cols+k] * b[k*b_cols+i]);
	//printf("*************");
	//printf("%lf\n" ,ALPHA * a[bix*a_cols+k] * b[k*b_cols+i]);
			// printf("%lf\t%lf\n" ,ALPHA * a[bix*a_cols+k], b[k*b_cols+i]);
        }
        c[bix*b_cols+i] = sum;   
           // printf("--------------------%lf-------------------\n\n\n",sum);
    }
    
    }
    
   
}

float matrixMul(int TA, int TB, int M, int N, int K, float ALPHA, 
        float *A_gpu, int lda, 
        float *B_gpu, int ldb,
        float BETA,
        float *C_gpu, int ldc, int b_bit){
/*
if((ALPHA !=1 && BETA!=0) || (ALPHA !=1 && BETA !=1)){
	printf("--------------------------------------------------------------------%lf,%lf\n",ALPHA,BETA);
	}
*/
float *a_device;

float *b_device;

float *result_device;


//srand(0);



hipMalloc((void**)&a_device,sizeof(float) *M * K);
hipMalloc((void**)&b_device,sizeof(float) *N * K);
hipMalloc((void**)&result_device,sizeof(float) *M * N);
hipMemcpy(a_device,A_gpu,sizeof(float) *M * K,hipMemcpyHostToDevice);
hipMemcpy(b_device,B_gpu,sizeof(float) *N * K,hipMemcpyHostToDevice);

/*
hipEvent_t start_device, stop_device;
float time_device;
hipEventCreate(&start_device);
hipEventCreate(&stop_device);
hipEventRecord( start_device, 0 );

	
int NN = 1024;
if(NN < 32){
    NN = 32;
}else if(NN<256){
	NN=256;}else if(NN<512){
			NN=512;}else{
					NN=1024;}
*/	
//b_bit=10;
//printf("%d\n",b_bit);   
int NN=N;
if(NN > 1024){
	NN = 1024;
	}				

dim3 gridsize(M,1,1);
dim3 blocksize(NN,1,1);
MatrixMul_device<<<gridsize,blocksize>>>(TA,TB,ALPHA,BETA,a_device,M,K,b_device,K,N,result_device,b_bit);

/*
hipEventRecord( stop_device, 0 );
hipEventSynchronize( stop_device );
hipEventElapsedTime( &time_device, start_device, stop_device );
hipEventDestroy( start_device );
hipEventDestroy( stop_device );
*/
//
hipMemcpy(C_gpu, result_device,sizeof(float) *M * N,hipMemcpyDeviceToHost);


hipFree(a_device);
hipFree(b_device);
hipFree(result_device);
//clock_t start_host = clock();


    
    return 0;
}
